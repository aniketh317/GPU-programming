#include "hip/hip_runtime.h"
/*This handles the block edge case too*/
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include "simpleLinearBlurFilter.hpp"

/*
 * CUDA Kernel Device code
 *
 */

__global__ void applySimpleLinearBlurFilter(uchar *r, uchar *g, uchar *b)
{
    // Consider using shared memory for the purpose of keeping the original input values
    // You can also use a constant array for handling edge cases or applying a custom filter

    __shared__ uchar r_old[16][16], g_old[16][16], b_old[16][16];

    int y = blockDim.x*blockIdx.x+threadIdx.x;
    int x = blockDim.y*blockIdx.y+threadIdx.y;

    if(x<d_columns && y<d_rows)
    {
        // When using shared memory you should store pixel values relevant to the current thread in a variable
        r_old[threadIdx.x][threadIdx.y] = r[d_columns*y+x];
        g_old[threadIdx.x][threadIdx.y] = g[d_columns*y+x];
        b_old[threadIdx.x][threadIdx.y] = b[d_columns*y+x];
    }
        // sync threads so that you can alter RGB values without causing race condition
    __syncthreads();
    // Apply a simple filter that averages the RGB values to the left and right of the pixel at the current thread id location
    if(((threadIdx.y)%16)!=0 && x<d_columns-1 && ((threadIdx.y+1)%16)!=0)
    {
        r[d_columns*y+x] = ((int)r_old[threadIdx.x][threadIdx.y-1]+(int)r_old[threadIdx.x][threadIdx.y]+(int)r_old[threadIdx.x][threadIdx.y+1])/3;
        g[d_columns*y+x] = ((int)g_old[threadIdx.x][threadIdx.y-1]+(int)g_old[threadIdx.x][threadIdx.y]+(int)g_old[threadIdx.x][threadIdx.y+1])/3;
        b[d_columns*y+x] = ((int)b_old[threadIdx.x][threadIdx.y-1]+(int)b_old[threadIdx.x][threadIdx.y]+(int)b_old[threadIdx.x][threadIdx.y+1])/3;
    }
    // Another area for improvement is handling when the current thread is at the let or right edge of the imput image
    else if(x>0 && x<d_columns-1 && (threadIdx.y+1)%16==0 && y<d_rows)
    {
        int id = blockIdx.y;
        //When pixel at right edge of the block
        r[d_columns*y+x] = ((int)r_old[threadIdx.x][threadIdx.y-1]+(int)r_old[threadIdx.x][threadIdx.y]+(int)r_e_even[y][id])/3;
        g[d_columns*y+x] = ((int)g_old[threadIdx.x][threadIdx.y-1]+(int)g_old[threadIdx.x][threadIdx.y]+(int)g_e_even[y][id])/3;
        b[d_columns*y+x] = ((int)b_old[threadIdx.x][threadIdx.y-1]+(int)b_old[threadIdx.x][threadIdx.y]+(int)b_e_even[y][id])/3;
    }
    else if(x>0 && x<d_columns-1 && (threadIdx.y)%16==0 && y<d_rows)
    {
        int id = blockIdx.y;
        //When pixel at left edge of the block
        r[d_columns*y+x] = ((int)r_old[threadIdx.x][threadIdx.y]+(int)r_old[threadIdx.x][threadIdx.y+1]+(int)r_e_odd[y][id])/3;
        g[d_columns*y+x] = ((int)g_old[threadIdx.x][threadIdx.y]+(int)g_old[threadIdx.x][threadIdx.y+1]+(int)g_e_odd[y][id])/3;
        b[d_columns*y+x] = ((int)b_old[threadIdx.x][threadIdx.y]+(int)b_old[threadIdx.x][threadIdx.y+1]+(int)b_e_odd[y][id])/3;
    }
}

__host__ float compareColorImages(uchar *r0, uchar *g0, uchar *b0, uchar *r1, uchar *g1, uchar *b1, int rows, int columns)
{
    cout << "Comparing actual and test pixel arrays\n";
    int numImagePixels = rows * columns;
    int imagePixelDifference = 0.0;

    for(int r = 0; r < rows; ++r)
    {
        for(int c = 0; c < columns; ++c)
        {
            uchar image0R = r0[r*columns+c];
            uchar image0G = g0[r*columns+c];
            uchar image0B = b0[r*columns+c];
            uchar image1R = r1[r*columns+c];
            uchar image1G = g1[r*columns+c];
            uchar image1B = b1[r*columns+c];
            imagePixelDifference += ((abs(image0R - image1R) + abs(image0G - image1G) + abs(image0B - image1B))/3);
        }
    }

    float meanImagePixelDifference = imagePixelDifference / numImagePixels;
    float scaledMeanDifferencePercentage = (meanImagePixelDifference / 255);
    printf("meanImagePixelDifference: %f scaledMeanDifferencePercentage: %f\n", meanImagePixelDifference, scaledMeanDifferencePercentage);
    return scaledMeanDifferencePercentage;
}

__host__ void allocateDeviceMemory(int rows, int columns)
{

    //Allocate device constant symbols for rows and columns
    hipMemcpyToSymbol(HIP_SYMBOL(d_rows), &rows, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_columns), &columns, sizeof(int), 0, hipMemcpyHostToDevice);
}

__host__ void executeKernel(uchar *r, uchar *g, uchar *b, int rows, int columns, int threadsPerBlock)
{
    cout << "Executing kernel\n";
    //Launch the convert CUDA Kernel

    int gridDimx = (columns>>4)+1; //Number of blocks along x
    int gridDimy = (rows>>4)+1; //Number of blocks along y

    dim3 grid(gridDimy, gridDimx); //grid dimension
    dim3 block(16, 16); //block dimension
    
    uchar **hr_e, **hg_e, **hb_e;
    
    uchar **hr_e_host, **hg_e_host, **hb_e_host;
    hr_e_host = (uchar **)malloc(rows*sizeof(uchar*));
    hg_e_host = (uchar **)malloc(rows*sizeof(uchar*));
    hb_e_host = (uchar **)malloc(rows*sizeof(uchar*));

    /*For odd allocation*/
    hipMalloc(&hr_e, rows*sizeof(uchar*));
    hipMalloc(&hg_e, rows*sizeof(uchar*));
    hipMalloc(&hb_e, rows*sizeof(uchar*));
    for(int i=0;i<rows;i++)
    {
        hipMalloc(&hr_e_host[i], (gridDimx)*sizeof(uchar));
        hipMalloc(&hg_e_host[i], (gridDimx)*sizeof(uchar));
        hipMalloc(&hb_e_host[i], (gridDimx)*sizeof(uchar));
        uchar *temp_r, *temp_g, *temp_b;
        temp_r = (uchar *)malloc((gridDimx)*sizeof(uchar));
        temp_g = (uchar *)malloc((gridDimx)*sizeof(uchar));
        temp_b = (uchar *)malloc((gridDimx)*sizeof(uchar));
        for(int j=1;j<=gridDimx;j++)
        {
            int x = 16*j-1;
            if(x<columns)
            {
                temp_r[j-1] = r[i*columns+x];
                temp_g[j-1] = g[i*columns+x];
                temp_b[j-1] = b[i*columns+x];
            }
        }
        hipMemcpy(hr_e_host[i], temp_r, gridDimx*sizeof(uchar), hipMemcpyHostToDevice);
        hipMemcpy(hg_e_host[i], temp_g, gridDimx*sizeof(uchar), hipMemcpyHostToDevice);
        hipMemcpy(hb_e_host[i], temp_b, gridDimx*sizeof(uchar), hipMemcpyHostToDevice);
        free(temp_r);
        free(temp_g);
        free(temp_b);
    }

    hipMemcpy(hr_e, hr_e_host, rows * sizeof(uchar*), hipMemcpyHostToDevice);
    hipMemcpy(hg_e, hg_e_host, rows * sizeof(uchar*), hipMemcpyHostToDevice);
    hipMemcpy(hb_e, hb_e_host, rows * sizeof(uchar*), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(r_e_odd), &hr_e, sizeof(uchar **));
    hipMemcpyToSymbol(HIP_SYMBOL(g_e_odd), &hg_e, sizeof(uchar **));
    hipMemcpyToSymbol(HIP_SYMBOL(b_e_odd), &hb_e, sizeof(uchar **));

    /*For even allocation*/
    hipMalloc(&hr_e, rows*sizeof(uchar*));
    hipMalloc(&hg_e, rows*sizeof(uchar*));
    hipMalloc(&hb_e, rows*sizeof(uchar*));
    for(int i=0;i<rows;i++)
    {
        hipMalloc(&hr_e_host[i], (gridDimx)*sizeof(uchar));
        hipMalloc(&hg_e_host[i], (gridDimx)*sizeof(uchar));
        hipMalloc(&hb_e_host[i], (gridDimx)*sizeof(uchar));
        uchar *temp_r, *temp_g, *temp_b;
        temp_r = (uchar *)malloc((gridDimx)*sizeof(uchar));
        temp_g = (uchar *)malloc((gridDimx)*sizeof(uchar));
        temp_b = (uchar *)malloc((gridDimx)*sizeof(uchar));
        for(int j=1;j<=gridDimx;j++)
        {
            int x = 16*j;
            if(x<columns)
            {
                temp_r[j-1] = r[i*columns+x];
                temp_g[j-1] = g[i*columns+x];
                temp_b[j-1] = b[i*columns+x];
            }
        }
        hipMemcpy(hr_e_host[i], temp_r, gridDimx*sizeof(uchar), hipMemcpyHostToDevice);
        hipMemcpy(hg_e_host[i], temp_g, gridDimx*sizeof(uchar), hipMemcpyHostToDevice);
        hipMemcpy(hb_e_host[i], temp_b, gridDimx*sizeof(uchar), hipMemcpyHostToDevice);
        free(temp_r);
        free(temp_g);
        free(temp_b);
    }
    hipMemcpy(hr_e, hr_e_host, rows * sizeof(uchar*), hipMemcpyHostToDevice);
    hipMemcpy(hg_e, hg_e_host, rows * sizeof(uchar*), hipMemcpyHostToDevice);
    hipMemcpy(hb_e, hb_e_host, rows * sizeof(uchar*), hipMemcpyHostToDevice);
    
    hipMemcpyToSymbol(HIP_SYMBOL(r_e_even), &hr_e, sizeof(uchar **));
    hipMemcpyToSymbol(HIP_SYMBOL(g_e_even), &hg_e, sizeof(uchar **));
    hipMemcpyToSymbol(HIP_SYMBOL(b_e_even), &hb_e, sizeof(uchar **));

    // Free the host-side arrays
    free(hr_e_host);
    free(hg_e_host);
    free(hb_e_host);
    hipError_t errBefore = hipGetLastError();
    if (errBefore != hipSuccess) 
    {
        std::cerr << "Error before kernel launch: " << hipGetErrorString(errBefore) << std::endl;
    }
    applySimpleLinearBlurFilter<<<grid, block>>>(r, g, b);
    hipError_t errSync = hipDeviceSynchronize();  // Wait for the kernel to finish
    if (errSync != hipSuccess) 
    {
        std::cerr << "Synchronization error: " << hipGetErrorString(errSync) << std::endl;
    }
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    cout << "Cleaning CUDA device\n";
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple<std::string, std::string, std::string, int> parseCommandLineArguments(int argc, char *argv[])
{
    cout << "Parsing CLI arguments\n";
    int threadsPerBlock = 256;
    std::string inputImage = "sloth.png";
    std::string outputImage = "grey-sloth.png";
    std::string currentPartId = "test";

    for (int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if (option.compare("-i") == 0)
        {
            inputImage = value;
        }
        else if (option.compare("-o") == 0)
        {
            outputImage = value;
        }
        else if (option.compare("-t") == 0)
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if (option.compare("-p") == 0)
        {
            currentPartId = value;
        }
    }
    cout << "inputImage: " << inputImage << " outputImage: " << outputImage << " currentPartId: " << currentPartId << " threadsPerBlock: " << threadsPerBlock << "\n";
    return {inputImage, outputImage, currentPartId, threadsPerBlock};
}

__host__ std::tuple<int, int, uchar *, uchar *, uchar *> readImageFromFile(std::string inputFile)
{
    cout << "Reading Image From File\n";
    Mat img = imread(inputFile, IMREAD_COLOR);
    
    const int rows = img.rows;
    const int columns = img.cols;
    size_t size = sizeof(uchar) * rows * columns;

    cout << "Rows: " << rows << " Columns: " << columns << "\n";

    uchar *r, *g, *b;
    hipMallocManaged(&r, size);
    hipMallocManaged(&g, size);
    hipMallocManaged(&b, size);
    
    for(int y = 0; y < rows; ++y)
    {
        for(int x = 0; x < columns; ++x)
        {
            Vec3b rgb = img.at<Vec3b>(y, x);
            b[y*columns+x] = rgb.val[0];
            g[y*columns+x]= rgb.val[1];
            r[y*columns+x] = rgb.val[2];
        }
    }

    return {rows, columns, r, g, b};
}

__host__ std::tuple<uchar *, uchar *, uchar *>applyBlurKernel(std::string inputImage)
{
    cout << "CPU applying kernel\n";
    Mat img = imread(inputImage, IMREAD_COLOR);
    const int rows = img.rows;
    const int columns = img.cols;

    uchar *r = (uchar *)malloc(sizeof(uchar) * rows * columns);
    uchar *g = (uchar *)malloc(sizeof(uchar) * rows * columns);
    uchar *b = (uchar *)malloc(sizeof(uchar) * rows * columns);

    for(int y = 0; y < rows; ++y)
    {
        for(int x = 1; x < columns-1; ++x)
        {
            Vec3b rgb0 = img.at<Vec3b>(y, x-1);
            Vec3b rgb1 = img.at<Vec3b>(y, x);
            Vec3b rgb2 = img.at<Vec3b>(y, x+1);
            b[y*columns+x] = (rgb0[0] + rgb1[0] + rgb2[0])/3;
            g[y*columns+x] = (rgb0[1] + rgb1[1] + rgb2[1])/3;
            r[y*columns+x] = (rgb0[2] + rgb1[2] + rgb2[2])/3;
        }
    }

    return {r, g, b};
}

int main(int argc, char *argv[])
{
    std::tuple<std::string, std::string, std::string, int> parsedCommandLineArgsTuple = parseCommandLineArguments(argc, argv);
    std::string inputImage = get<0>(parsedCommandLineArgsTuple);
    std::string outputImage = get<1>(parsedCommandLineArgsTuple);
    std::string currentPartId = get<2>(parsedCommandLineArgsTuple);
    int threadsPerBlock = get<3>(parsedCommandLineArgsTuple);
    try 
    {
        auto[rows, columns, r, g, b] = readImageFromFile(inputImage);
        allocateDeviceMemory(rows, columns);
        executeKernel(r, g, b, rows, columns, threadsPerBlock);

        Mat colorImage(rows, columns, CV_8UC3);
        vector<int> compression_params;
        compression_params.push_back(IMWRITE_PNG_COMPRESSION);
        compression_params.push_back(9);

        for(int y = 0; y < rows; ++y)
        {
            for(int x = 0; x < columns; ++x)
            {
                colorImage.at<Vec3b>(y,x) = Vec3b(b[y*columns+x], g[y*columns+x], r[y*columns+x]);
            }
        }

        imwrite(outputImage, colorImage, compression_params);

        auto[test_r, test_g, test_b] = applyBlurKernel(inputImage);
        
        float scaledMeanDifferencePercentage = compareColorImages(r, g, b, test_r, test_g, test_b, rows, columns) * 100;
        cout << "Mean difference percentage: " << scaledMeanDifferencePercentage << "\n";

        cleanUpDevice();
    }
    catch (Exception &error_)
    {
        cout << "Caught exception: " << error_.what() << endl;
        return 1;
    }
    return 0;
}